#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>

__device__ double G = 6.673e-11;

__global__ void inner_cal_ef0(double pos1x, double pos1y, double pos1z, double m1 , double *mass, double *posx, double *posy, double *posz, double *forx, double *fory, double *forz, int i)
{
    int j = threadIdx.x + blockIdx.x * blockDim.x;
    if ( i != j )
    {
        double pos2x = posx[j],
	       pos2y = posy[j],
	       pos2z = posz[j];
	double m2 = mass[j];
	double relposx, relposy, relposz;
	double univecx, univecy, univecz;
	double magpos2, magpos;
	relposx = pos1x - pos2x;
	relposy = pos1y - pos2y;
	relposz = pos1z - pos2z;
	magpos2 = relposx * relposx + relposy * relposy + relposz * relposz;
	magpos  = sqrt(magpos2);
	univecx = relposx / magpos;
	univecy = relposy / magpos;
	univecz = relposz / magpos;
	magpos  = - G * m1 * m2 / magpos2;
	forx[i] += magpos * univecx;
	fory[i] += magpos * univecy;
	forz[i] += magpos * univecz;
    }
}

// simple kernel function that adds two vectors
__global__ void calculate(double *mass, double *posx, double *posy, double *posz, double *forx, double *fory, double *forz, int ef_type, double ef_mass, int N)
{
    if ( N == 1 )
    {
        if ( ef_type == 1 )
        {
	    double m = mass[0];
            double relposx = posx[0], relposy = posy[0], relposz = posz[0];
	    double univecx, univecy, univecz;
	    double magpos2, magpos;
	    magpos2 = relposx * relposx + relposy * relposy + relposz * relposz;
	    magpos = sqrt(magpos2);
	    univecx = relposx / magpos;
	    univecy = relposy / magpos;
	    univecz = relposz / magpos;
	    magpos = - G * ef_mass * m / magpos2;
	    forx[0] = magpos * univecx;
	    fory[0] = magpos * univecy;
	    forz[0] = magpos * univecz;
	}
    }
    else
    {
	    int i = threadIdx.x + blockIdx.x * blockDim.x;
	    int blocks = 1;
	    double pos1x = posx[i], pos1y = posy[i], pos1z = posz[i];
	    double m1 = mass[i];
	    inner_cal_ef0<<< blocks, N >>>(pos1x, pos1y, pos1z, m1, mass, posx, posy, posz, forx, fory, forz, i);
	    hipDeviceSynchronize();
    }
}
// function called from main fortran program
extern "C" void KERNEL_WRAPPER(int *ef_type_p, double *ef_mass_p, int *N_p, double *mass, double *posx, double *posy, double *posz, double *forx, double *fory, double *forz)
{
    double *mass_d; // declare GPU vector copies
    double *posx_d, *posy_d, *posz_d;
    double *forx_d, *fory_d, *forz_d;

    int blocks = 1;
    int ef_type = *ef_type_p;
    double ef_mass = *ef_mass_p;
    int N = *N_p;

    // Allocate memory on GPU
    hipMalloc( (void **)&mass_d, sizeof(double) * N );
    hipMalloc( (void **)&posx_d, sizeof(double) * N );
    hipMalloc( (void **)&posy_d, sizeof(double) * N );
    hipMalloc( (void **)&posz_d, sizeof(double) * N );
    hipMalloc( (void **)&forx_d, sizeof(double) * N );
    hipMalloc( (void **)&fory_d, sizeof(double) * N );
    hipMalloc( (void **)&forz_d, sizeof(double) * N );

    // copy vectors from CPU to GPU
    hipMemcpy( mass_d, mass, sizeof(double) * N, hipMemcpyHostToDevice );
    hipMemcpy( posx_d, posx, sizeof(double) * N, hipMemcpyHostToDevice );
    hipMemcpy( posy_d, posy, sizeof(double) * N, hipMemcpyHostToDevice );
    hipMemcpy( posz_d, posz, sizeof(double) * N, hipMemcpyHostToDevice );
    hipMemcpy( forx_d, forx, sizeof(double) * N, hipMemcpyHostToDevice );
    hipMemcpy( fory_d, fory, sizeof(double) * N, hipMemcpyHostToDevice );
    hipMemcpy( forz_d, forz, sizeof(double) * N, hipMemcpyHostToDevice );

    // call function on GPU
    calculate<<< blocks, N >>>(mass_d, posx_d, posy_d, posz_d, forx_d, fory_d, forz_d, ef_type, ef_mass, N);

    hipDeviceSynchronize();

    // copy vectors back from GPU to CPU
    hipMemcpy( forx, forx_d, sizeof(double) * N, hipMemcpyDeviceToHost );
    hipMemcpy( fory, fory_d, sizeof(double) * N, hipMemcpyDeviceToHost );
    hipMemcpy( forz, forz_d, sizeof(double) * N, hipMemcpyDeviceToHost );

    // free GPU memory
    hipFree(mass_d);
    hipFree(posx_d);
    hipFree(posy_d);
    hipFree(posz_d);
    hipFree(forx_d);
    hipFree(fory_d);
    hipFree(forz_d);
    return;
}
